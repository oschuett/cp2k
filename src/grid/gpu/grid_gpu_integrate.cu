
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2020 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: GPL-2.0-or-later                                 */
/*----------------------------------------------------------------------------*/

#ifdef __GRID_CUDA

#include <algorithm>
#include <assert.h>
#include <cuda.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 0
#include "../common/grid_common.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_integrate.h"

/*******************************************************************************
 * \brief Cuda kernel for integrating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_kernel(const kernel_params params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  fill_smem_task(&params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params.smem_cab_offset];
  double *smem_alpha = &shared_memory[params.smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params.smem_cxyz_offset];

  cxyz_to_grid(&params, &task, smem_cxyz, params.grid);
  compute_alpha(&params, &task, smem_alpha);
  cab_to_cxyz(&params, &task, smem_alpha, smem_cab, smem_cxyz);

  //  block_to_cab<IS_FUNC_AB>(params, &task, smem_cab);
  //
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that integrates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_integrate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool orthorhombic, const bool compute_tau,
    const bool calculate_forces, const int npts_global[3],
    const int npts_local[3], const int shift_local[3],
    const int border_width[3], const double dh[3][3], const double dh_inv[3][3],
    const cudaStream_t stream, const double *pab_blocks_dev,
    const double *grid_dev, double *hab_blocks_dev, double *forces_dev,
    double *virial_dev) {

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  init_constant_memory();

  // Compute required shared memory.
  // TODO: Currently, cab's indicies run over 0...ncoset[lmax],
  //       however only ncoset(lmin)...ncoset(lmax) are actually needed.
  const int lmax = task_list->lmax;
  const int lp_max = 2 * lmax;
  const int cab_len = ncoset(lmax) * ncoset(lmax);
  const int alpha_len = 3 * (lmax + 1) * (lmax + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const int alpha_cxyz_len = alpha_len + cxyz_len;
  const size_t smem_per_block = (cab_len + alpha_cxyz_len) * sizeof(double);

  if (smem_per_block > 48 * 1024) {
    fprintf(stderr, "ERROR: Not enough shared memory.\n");
    fprintf(stderr, "alpha_len: %i, ", alpha_len);
    fprintf(stderr, "cxyz_len: %i, ", alpha_cxyz_len);
    fprintf(stderr, "cab_len: %i, ", cab_len);
    fprintf(stderr, "total smem_per_block: %f kb\n\n", smem_per_block / 1024.0);
    abort();
  }

  // kernel parameters
  kernel_params params;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.orthorhombic = orthorhombic;
  params.compute_tau = compute_tau;
  params.calculate_forces = calculate_forces;
  params.grid = grid_dev;
  params.tasks = task_list->tasks_dev;
  params.atom_kinds = task_list->atom_kinds_dev;
  params.basis_sets = task_list->basis_sets_dev;
  params.block_offsets = task_list->block_offsets_dev;
  params.atom_positions = task_list->atom_positions_dev;
  params.pab_blocks = pab_blocks_dev;
  params.hab_blocks = hab_blocks_dev;
  params.forces = forces_dev;
  params.virial = virial_dev;
  memcpy(params.dh, dh, 9 * sizeof(double));
  memcpy(params.dh_inv, dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, shift_local, 3 * sizeof(int));
  memcpy(params.border_width, border_width, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 8, 8);

  integrate_kernel<<<nblocks, threads_per_block, smem_per_block, stream>>>(
      params);
}

#endif // __GRID_CUDA
// EOF
