
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2020 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: GPL-2.0-or-later                                 */
/*----------------------------------------------------------------------------*/

#ifdef __GRID_CUDA

#include <algorithm>
#include <assert.h>
#include <cuda.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 1
#include "../common/grid_common.h"
#include "../common/grid_prepare_pab.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_collocate.h"

/*******************************************************************************
 * \brief Adds given value to matrix element cab[idx(b)][idx(a)].
 * \author Ole Schuett
 ******************************************************************************/
__device__ static inline void prep_term(const orbital a, const orbital b,
                                        const double value, const int n,
                                        double *cab) {
  atomicAddDouble(&cab[idx(b) * n + idx(a)], value);
}

// /*******************************************************************************
//  * \brief Transforms coefficients C_ab into C_xyz.
//  * \author Ole Schuett
//  ******************************************************************************/
// __device__ static void cab_to_cxyz(const kernel_params *params,
//                                    const smem_task *task, const double
//                                    *alpha, GRID_CONST_WHEN_COLLOCATE double
//                                    *cab, GRID_CONST_WHEN_INTEGRATE double
//                                    *cxyz) {
//
//   //   *** initialise the coefficient matrix, we transform the sum
//   //
//   // sum_{lxa,lya,lza,lxb,lyb,lzb} P_{lxa,lya,lza,lxb,lyb,lzb} *
//   //         (x-a_x)**lxa (y-a_y)**lya (z-a_z)**lza (x-b_x)**lxb (y-a_y)**lya
//   //         (z-a_z)**lza
//   //
//   // into
//   //
//   // sum_{lxp,lyp,lzp} P_{lxp,lyp,lzp} (x-p_x)**lxp (y-p_y)**lyp (z-p_z)**lzp
//   //
//   // where p is center of the product gaussian, and lp = la_max + lb_max
//   // (current implementation is l**7)
//
//   // strides for accessing alpha
//   const int s3 = (task->lp + 1);
//   const int s2 = (task->la_max + 1) * s3;
//   const int s1 = (task->lb_max + 1) * s2;
//
//   // TODO: Maybe we can transpose alpha to index it directly with ico and
//   jco. for (int lzp = threadIdx.z; lzp <= task->lp; lzp += blockDim.z) {
//     for (int lyp = threadIdx.y; lyp <= task->lp - lzp; lyp += blockDim.y) {
//       for (int lxp = threadIdx.x; lxp <= task->lp - lzp - lyp;
//            lxp += blockDim.x) {
//
//         double reg = 0.0; // accumulate into a register
//         for (int jco = 0; jco < ncoset(task->lb_max); jco++) {
//           const orbital b = coset_inv[jco];
//           for (int ico = 0; ico < ncoset(task->la_max); ico++) {
//             const orbital a = coset_inv[ico];
//
//             const double p = task->prefactor *
//                              alpha[0 * s1 + b.l[0] * s2 + a.l[0] * s3 + lxp]
//                              * alpha[1 * s1 + b.l[1] * s2 + a.l[1] * s3 +
//                              lyp] * alpha[2 * s1 + b.l[2] * s2 + a.l[2] * s3
//                              + lzp];
//             const int cab_index = jco * task->n1 + ico; // [jco, ico]
//             reg += p * cab[cab_index];
//           }
//         }
//         cxyz[coset(lxp, lyp, lzp)] = reg; // overwrite - no zeroing needed.
//       }
//     }
//   }
//   __syncthreads(); // because of concurrent writes to cxyz
// }
//
/*******************************************************************************
 * \brief Decontracts the subblock, going from spherical to cartesian harmonics.
 * \author Ole Schuett
 ******************************************************************************/
template <bool IS_FUNC_AB>
__device__ static void block_to_cab(const kernel_params *params,
                                    const smem_task *task, double *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // Zero cab.
  if (threadIdx.z == 0) {
    for (int i = threadIdx.y; i < task->n2; i += blockDim.y) {
      for (int j = threadIdx.x; j < task->n1; j += blockDim.x) {
        cab[i * task->n1 + j] = 0.0;
      }
    }
  }
  __syncthreads(); // because of concurrent writes to cab

  // Decontract block, apply prepare_pab, and store in cab.
  // This is a double matrix product. Since the pab block can be quite large the
  // two products are fused to conserve shared memory.
  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      double block_val;
      if (task->block_transposed) {
        block_val = task->pab_block[j * task->nsgfb + i];
      } else {
        block_val = task->pab_block[i * task->nsgfa + j];
      }

      if (IS_FUNC_AB) {
        // fast path for common case
        for (int k = threadIdx.z; k < task->ncosetb; k += blockDim.z) {
          const double sphib = task->sphib[i * task->maxcob + k];
          for (int l = 0; l < task->ncoseta; l++) {
            const double sphia = task->sphia[j * task->maxcoa + l];
            const double pab_val = block_val * sphia * sphib;
            atomicAddDouble(&cab[k * task->ncoseta + l], pab_val);
          }
        }
      } else {
        // Since prepare_pab is a register hog we use it only when really needed
        for (int k = threadIdx.z; k < task->ncosetb; k += blockDim.z) {
          const orbital b = coset_inv[k];
          for (int l = 0; l < task->ncoseta; l++) {
            const orbital a = coset_inv[l];
            const double sphia = task->sphia[j * task->maxcoa + idx(a)];
            const double sphib = task->sphib[i * task->maxcob + idx(b)];
            const double pab_val = block_val * sphia * sphib;
            prepare_pab(params->func, a, b, task->zeta, task->zetb, pab_val,
                        task->n1, cab);
          }
        }
      }
    }
  }
  __syncthreads(); // because of concurrent writes to cab
}

/*******************************************************************************
 * \brief Cuda kernel for collocating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
template <bool IS_FUNC_AB>
__device__ static void collocate_kernel(const kernel_params *params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  fill_smem_task(params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params->smem_cab_offset];
  double *smem_alpha = &shared_memory[params->smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params->smem_cxyz_offset];

  block_to_cab<IS_FUNC_AB>(params, &task, smem_cab);

  compute_alpha(params, &task, smem_alpha);
  cab_to_cxyz(params, &task, smem_alpha, smem_cab, smem_cxyz);
  cxyz_to_grid(params, &task, smem_cxyz, params->grid);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel that can only collocate GRID_FUNC_AB.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void collocate_kernel_density(const kernel_params params) {
  collocate_kernel<true>(&params);
}

/*******************************************************************************
 * \brief Cuda kernel that can collocate any function, ie. GRID_FUNC_*.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void collocate_kernel_anyfunc(const kernel_params params) {
  collocate_kernel<false>(&params);
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that collocates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_collocate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool orthorhombic, const enum grid_func func,
    const int npts_global[3], const int npts_local[3], const int shift_local[3],
    const int border_width[3], const double dh[3][3], const double dh_inv[3][3],
    const cudaStream_t stream, const double *pab_blocks_dev, double *grid_dev) {

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  init_constant_memory();

  // Compute required shared memory.
  // TODO: Currently, cab's indicies run over 0...ncoset[lmax],
  //       however only ncoset(lmin)...ncoset(lmax) are actually needed.
  const prepare_ldiffs ldiffs = prepare_get_ldiffs(func);
  const int la_max = task_list->lmax + ldiffs.la_max_diff;
  const int lb_max = task_list->lmax + ldiffs.lb_max_diff;
  const int lp_max = la_max + lb_max;
  const int cab_len = ncoset(lb_max) * ncoset(la_max);
  const int alpha_len = 3 * (lb_max + 1) * (la_max + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const size_t smem_per_block =
      (cab_len + alpha_len + cxyz_len) * sizeof(double);

  if (smem_per_block > 48 * 1024) {
    fprintf(stderr, "ERROR: Not enough shared memory.\n");
    fprintf(stderr, "cab_len: %i, ", cab_len);
    fprintf(stderr, "alpha_len: %i, ", alpha_len);
    fprintf(stderr, "cxyz_len: %i, ", cxyz_len);
    fprintf(stderr, "total smem_per_block: %f kb\n\n", smem_per_block / 1024.0);
    abort();
  }

  // kernel parameters
  kernel_params params;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.orthorhombic = orthorhombic;
  params.func = func;
  params.grid = grid_dev;
  params.la_min_diff = ldiffs.la_min_diff;
  params.lb_min_diff = ldiffs.lb_min_diff;
  params.la_max_diff = ldiffs.la_max_diff;
  params.lb_max_diff = ldiffs.lb_max_diff;
  params.tasks = task_list->tasks_dev;
  params.atom_kinds = task_list->atom_kinds_dev;
  params.basis_sets = task_list->basis_sets_dev;
  params.block_offsets = task_list->block_offsets_dev;
  params.atom_positions = task_list->atom_positions_dev;
  params.pab_blocks = pab_blocks_dev;
  memcpy(params.dh, dh, 9 * sizeof(double));
  memcpy(params.dh_inv, dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, shift_local, 3 * sizeof(int));
  memcpy(params.border_width, border_width, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 8, 8);

  if (func == GRID_FUNC_AB) {
    collocate_kernel_density<<<nblocks, threads_per_block, smem_per_block,
                               stream>>>(params);
  } else {
    collocate_kernel_anyfunc<<<nblocks, threads_per_block, smem_per_block,
                               stream>>>(params);
  }
}

#endif // __GRID_CUDA
// EOF
