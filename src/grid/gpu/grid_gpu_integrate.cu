
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2020 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: GPL-2.0-or-later                                 */
/*----------------------------------------------------------------------------*/

#ifdef __GRID_CUDA

#include <algorithm>
#include <assert.h>
#include <cuda.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 0
#include "../common/grid_common.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_integrate.h"

/*******************************************************************************
 * \brief Decontracts the subblock, going from spherical to cartesian harmonics.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void store_hab(const kernel_params *params,
                                 const smem_task *task, const double *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // This is a double matrix product. Since the block can be quite large the
  // two products are fused to conserve shared memory.
  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      for (int k = threadIdx.z; k < task->ncosetb; k += blockDim.z) {
        double block_val = 0.0;
        const double sphib = task->sphib[i * task->maxcob + k];
        for (int l = 0; l < task->ncoseta; l++) {
          const double sphia = task->sphia[j * task->maxcoa + l];
          block_val += cab[k * task->ncoseta + l] * sphia * sphib;
        }
        if (task->block_transposed) {
          atomicAddDouble(&task->hab_block[j * task->nsgfb + i], block_val);
        } else {
          atomicAddDouble(&task->hab_block[i * task->nsgfa + j], block_val);
        }
      }
    }
  }
  __syncthreads(); // TODO: not really neded because of concurrent writes to
  // cab
}

/*******************************************************************************
 * \brief Cuda kernel for integrating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_kernel(const kernel_params params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  fill_smem_task(&params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params.smem_cab_offset];
  double *smem_alpha = &shared_memory[params.smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params.smem_cxyz_offset];

  memset(smem_cxyz, 0, ncoset(task.lp) * sizeof(double));
  __syncthreads();

  cxyz_to_grid(&params, &task, smem_cxyz, params.grid);

  memset(smem_cab, 0, task.n1_cab * task.n2_cab * sizeof(double));
  __syncthreads();

  compute_alpha(&params, &task, smem_alpha);
  cab_to_cxyz(&params, &task, smem_alpha, smem_cab, smem_cxyz);

  store_hab(&params, &task, smem_cab);

  // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
  //  printf("la_min: %i %lb_min: %i, ncoset: %i, %i \n",
  //      task.la_min, task.lb_min,
  //      ncoset(task.la_min-1), ncoset(task.lb_min-1));
  //  //    for (int k = 0; k < task.ncosetb; k++) {
  //  //      for (int l = 0; l < task.ncoseta; l++) {
  //  //          printf("cab %i %i %le\n", k, l, smem_cab[k * task.ncoseta +
  //  l]);
  //  //      }
  //  //    }
  //  // printf("cxyz %i %i %le\n",0, 0, smem_cxyz[0]);
  //}
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that integrates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_integrate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool orthorhombic, const bool compute_tau,
    const bool calculate_forces, const int npts_global[3],
    const int npts_local[3], const int shift_local[3],
    const int border_width[3], const double dh[3][3], const double dh_inv[3][3],
    const cudaStream_t stream, const double *pab_blocks_dev,
    const double *grid_dev, double *hab_blocks_dev, double *forces_dev,
    double *virial_dev) {

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  init_constant_memory();

  // Compute required shared memory.
  // TODO: Currently, cab's indicies run over 0...ncoset[lmax],
  //       however only ncoset(lmin)...ncoset(lmax) are actually needed.
  const int lmax = task_list->lmax;
  const int lp_max = 2 * lmax;
  const int cab_len = ncoset(lmax) * ncoset(lmax);
  const int alpha_len = 3 * (lmax + 1) * (lmax + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const size_t smem_per_block =
      (cab_len + alpha_len + cxyz_len) * sizeof(double);

  if (smem_per_block > 48 * 1024) {
    fprintf(stderr, "ERROR: Not enough shared memory.\n");
    fprintf(stderr, "cab_len: %i, ", cab_len);
    fprintf(stderr, "alpha_len: %i, ", alpha_len);
    fprintf(stderr, "cxyz_len: %i, ", cxyz_len);
    fprintf(stderr, "total smem_per_block: %f kb\n\n", smem_per_block / 1024.0);
    abort();
  }

  // kernel parameters
  kernel_params params;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.orthorhombic = orthorhombic;
  params.compute_tau = compute_tau;
  params.calculate_forces = calculate_forces;
  params.grid = grid_dev;
  params.tasks = task_list->tasks_dev;
  params.atom_kinds = task_list->atom_kinds_dev;
  params.basis_sets = task_list->basis_sets_dev;
  params.block_offsets = task_list->block_offsets_dev;
  params.atom_positions = task_list->atom_positions_dev;
  params.pab_blocks = pab_blocks_dev;
  params.hab_blocks = hab_blocks_dev;
  params.forces = forces_dev;
  params.virial = virial_dev;
  memcpy(params.dh, dh, 9 * sizeof(double));
  memcpy(params.dh_inv, dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, shift_local, 3 * sizeof(int));
  memcpy(params.border_width, border_width, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 8, 8);

  integrate_kernel<<<nblocks, threads_per_block, smem_per_block, stream>>>(
      params);
}

#endif // __GRID_CUDA
// EOF
