
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2020 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: GPL-2.0-or-later                                 */
/*----------------------------------------------------------------------------*/

#ifdef __GRID_CUDA

#include <algorithm>
#include <assert.h>
#include <cuda.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 0
#include "../common/grid_common.h"
#include "../common/grid_process_vab.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_integrate.h"

/*******************************************************************************
 * \brief Decontracts the subblock, going from spherical to cartesian harmonics.
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU>
__device__ static void store_hab(const kernel_params *params,
                                 const smem_task *task, const double *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // This is a double matrix product. Since the block can be quite large the
  // two products are fused to conserve shared memory.
  // TODO: move into smem_task
  const int ico_start =
      (task->la_min_basis > 0) ? ncoset(task->la_min_basis - 1) : 0;
  const int jco_start =
      (task->lb_min_basis > 0) ? ncoset(task->lb_min_basis - 1) : 0;

  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      for (int jco = jco_start + threadIdx.z; jco < ncoset(task->lb_max_basis);
           jco += blockDim.z) {
        const orbital b = coset_inv[jco];
        double block_val = 0.0;
        const double sphib = task->sphib[i * task->maxcob + jco];
        for (int ico = ico_start; ico < ncoset(task->la_max_basis); ico++) {
          const orbital a = coset_inv[ico];
          double hab = 0.0;
          if (COMPUTE_TAU) {
            // Since process_tau is a register hog we use it only when needed.
            hab = extract_hab_tau(a, b, task->zeta, task->zetb, task->n1, cab);
          } else {
            // fast path for common case
            hab = extract_hab(a, b, task->n1, cab);
          }

          const double sphia = task->sphia[j * task->maxcoa + ico];
          block_val += hab * sphia * sphib;
        }
        if (task->block_transposed) {
          atomicAddDouble(&task->hab_block[j * task->nsgfb + i], block_val);
        } else {
          atomicAddDouble(&task->hab_block[i * task->nsgfa + j], block_val);
        }
      }
    }
  }
  __syncthreads(); // TODO: Probably not needed
}

/*******************************************************************************
 * \brief TODO
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU>
__device__ static void store_forces_and_virial(const kernel_params *params,
                                               const smem_task *task,
                                               const double *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // Decontract block, apply prepare_pab, and store in cab.
  // This is a double matrix product. Since the pab block can be quite large the
  // two products are fused to conserve shared memory.

  // TODO: move into smem_task
  const int ico_start =
      (task->la_min_basis > 0) ? ncoset(task->la_min_basis - 1) : 0;
  const int jco_start =
      (task->lb_min_basis > 0) ? ncoset(task->lb_min_basis - 1) : 0;

  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      double block_val;
      if (task->block_transposed) {
        block_val = task->pab_block[j * task->nsgfb + i];
      } else {
        block_val = task->pab_block[i * task->nsgfa + j];
      }
      for (int jco = jco_start + threadIdx.z; jco < task->ncosetb;
           jco += blockDim.z) {
        const double sphib = task->sphib[i * task->maxcob + jco];
        for (int ico = ico_start; ico < task->ncoseta; ico++) {
          const double sphia = task->sphia[j * task->maxcoa + ico];
          const double pabval = block_val * sphia * sphib;
          const orbital b = coset_inv[jco];
          const orbital a = coset_inv[ico];
          for (int k = 0; k < 3; k++) {
            const double force_a =
                pabval * task->off_diagonals_twice *
                extract_force_a(a, b, k, task->zeta, task->n1, cab);
            atomicAddDouble(&task->forces_a[k], force_a);
            const double force_b =
                pabval * task->off_diagonals_twice *
                extract_force_b(a, b, k, task->zetb, task->rab, task->n1, cab);
            atomicAddDouble(&task->forces_b[k], force_b);
          }
          if (params->virial != NULL) {
            for (int k = 0; k < 3; k++) {
              for (int l = 0; l < 3; l++) {
                const double virial_a =
                    extract_virial_a(a, b, k, l, task->zeta, task->n1, cab);
                const double virial_b = extract_virial_b(
                    a, b, k, l, task->zetb, task->rab, task->n1, cab);
                const double virial =
                    pabval * task->off_diagonals_twice * (virial_a + virial_b);
                atomicAddDouble(&params->virial[k * 3 + l], virial);
              }
            }
          }
        }
      }
    }
  }
  __syncthreads(); // TODO: Probably not needed
}

/*******************************************************************************
 * \brief Cuda kernel for integrating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU, bool CALCULATE_FORCES>
__device__ static void integrate_kernel(const kernel_params *params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  fill_smem_task(params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params->smem_cab_offset];
  double *smem_alpha = &shared_memory[params->smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params->smem_cxyz_offset];

  memset(smem_cxyz, 0, ncoset(task.lp) * sizeof(double));
  __syncthreads();

  cxyz_to_grid(params, &task, smem_cxyz, params->grid);

  memset(smem_cab, 0, task.n1 * task.n2 * sizeof(double));
  __syncthreads();

  compute_alpha(params, &task, smem_alpha);
  cab_to_cxyz(params, &task, smem_alpha, smem_cab, smem_cxyz);

  store_hab<COMPUTE_TAU>(params, &task, smem_cab);

  if (CALCULATE_FORCES) {
    store_forces_and_virial<COMPUTE_TAU>(params, &task, smem_cab);
  }

  // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
  //  printf("la_min: %i %lb_min: %i, ncoset: %i, %i \n",
  //      task.la_min, task.lb_min,
  //      ncoset(task.la_min-1), ncoset(task.lb_min-1));
  //  //    for (int k = 0; k < task.ncosetb; k++) {
  //  //      for (int l = 0; l < task.ncoseta; l++) {
  //  //          printf("cab %i %i %le\n", k, l, smem_cab[k * task.ncoseta +
  //  l]);
  //  //      }
  //  //    }
  //  // printf("cxyz %i %i %le\n",0, 0, smem_cxyz[0]);
  //}
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=false & calculate_forces=false
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_density(const kernel_params params) {
  integrate_kernel<false, false>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=true & calculate_forces=false.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_tau(const kernel_params params) {
  integrate_kernel<true, false>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=false & calculate_forces=true.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_density_forces(const kernel_params params) {
  integrate_kernel<false, true>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=true & calculate_forces=true.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_tau_forces(const kernel_params params) {
  integrate_kernel<true, true>(&params);
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that integrates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_integrate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool orthorhombic, const bool compute_tau,
    const int npts_global[3], const int npts_local[3], const int shift_local[3],
    const int border_width[3], const double dh[3][3], const double dh_inv[3][3],
    const cudaStream_t stream, const double *pab_blocks_dev,
    const double *grid_dev, double *hab_blocks_dev, double *forces_dev,
    double *virial_dev) {

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  init_constant_memory();

  // Compute required shared memory.
  // TODO: Currently, cab's indicies run over 0...ncoset[lmax],
  //       however only ncoset(lmin)...ncoset(lmax) are actually needed.
  const bool calculate_forces = forces_dev != NULL;
  const bool calculate_virial = virial_dev != NULL;
  const process_ldiffs ldiffs =
      process_get_ldiffs(calculate_forces, calculate_virial, compute_tau);
  const int la_max = task_list->lmax + ldiffs.la_max_diff;
  const int lb_max = task_list->lmax + ldiffs.lb_max_diff;
  const int lp_max = la_max + lb_max;
  const int cab_len = ncoset(lb_max) * ncoset(la_max);
  const int alpha_len = 3 * (lb_max + 1) * (la_max + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const size_t smem_per_block =
      (cab_len + alpha_len + cxyz_len) * sizeof(double);

  if (smem_per_block > 48 * 1024) {
    fprintf(stderr, "ERROR: Not enough shared memory in grid_gpu_integrate.\n");
    fprintf(stderr, "cab_len: %i, ", cab_len);
    fprintf(stderr, "alpha_len: %i, ", alpha_len);
    fprintf(stderr, "cxyz_len: %i, ", cxyz_len);
    fprintf(stderr, "total smem_per_block: %f kb\n\n", smem_per_block / 1024.0);
    abort();
  }

  // assert(compute_tau == false);

  // kernel parameters
  kernel_params params;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.orthorhombic = orthorhombic;
  params.grid = grid_dev;
  params.tasks = task_list->tasks_dev;
  params.atom_kinds = task_list->atom_kinds_dev;
  params.basis_sets = task_list->basis_sets_dev;
  params.block_offsets = task_list->block_offsets_dev;
  params.atom_positions = task_list->atom_positions_dev;
  params.pab_blocks = pab_blocks_dev;
  params.hab_blocks = hab_blocks_dev;
  params.forces = forces_dev;
  params.virial = virial_dev;
  params.la_min_diff = ldiffs.la_min_diff;
  params.lb_min_diff = ldiffs.lb_min_diff;
  params.la_max_diff = ldiffs.la_max_diff;
  params.lb_max_diff = ldiffs.lb_max_diff;
  memcpy(params.dh, dh, 9 * sizeof(double));
  memcpy(params.dh_inv, dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, shift_local, 3 * sizeof(int));
  memcpy(params.border_width, border_width, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 8, 8);

  if (!compute_tau && !calculate_forces) {
    integrate_density<<<nblocks, threads_per_block, smem_per_block, stream>>>(
        params);
  } else if (compute_tau && !calculate_forces) {
    integrate_tau<<<nblocks, threads_per_block, smem_per_block, stream>>>(
        params);
  } else if (!compute_tau && calculate_forces) {
    integrate_density_forces<<<nblocks, threads_per_block, smem_per_block,
                               stream>>>(params);
  } else if (compute_tau && calculate_forces) {
    integrate_tau_forces<<<nblocks, threads_per_block, smem_per_block,
                           stream>>>(params);
  }
}

#endif // __GRID_CUDA
// EOF
