
#include <hip/hip_runtime.h>
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2020 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: GPL-2.0-or-later                                 */
/*----------------------------------------------------------------------------*/

#ifdef __GRID_CUDA

#include <algorithm>
#include <assert.h>
#include <cuda.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 0
#include "../common/grid_common.h"
#include "../common/grid_process_vab.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_integrate.h"

/*******************************************************************************
 * \brief Decontracts the subblock, going from spherical to cartesian harmonics.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void store_hab(const kernel_params *params,
                                 const smem_task *task, const double *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // This is a double matrix product. Since the block can be quite large the
  // two products are fused to conserve shared memory.
  const int ico_start =
      (task->la_min_basis > 0) ? ncoset(task->la_min_basis - 1) : 0;
  const int jco_start =
      (task->lb_min_basis > 0) ? ncoset(task->lb_min_basis - 1) : 0;

  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      for (int jco = jco_start + threadIdx.z; jco < ncoset(task->lb_max_basis);
           jco += blockDim.z) {
        const orbital b = coset_inv[jco];
        double block_val = 0.0;
        const double sphib = task->sphib[i * task->maxcob + jco];
        for (int ico = ico_start; ico < ncoset(task->la_max_basis); ico++) {
          const orbital a = coset_inv[ico];
          double habval = 0.0;
          const double ftza = 2.0 * task->zeta; // TODO precompute
          const double ftzb = 2.0 * task->zetb;
          if (params->compute_tau) {
            // TODO refactor into extract_normal_hab, extract_tau_hab
            // allso create extract_virial(i,j)  to extract only one value
            process_tau(a, b, ftza, ftzb, task->rab, task->n1, cab, NULL,
                        &habval, NULL, NULL);
          } else {
            process_normal(a, b, 1.0, ftza, ftzb, task->rab, task->n1, cab,
                           NULL, &habval, NULL, NULL);
          }

          const double sphia = task->sphia[j * task->maxcoa + ico];
          block_val += habval * sphia * sphib;
        }
        if (task->block_transposed) {
          atomicAddDouble(&task->hab_block[j * task->nsgfb + i], block_val);
        } else {
          atomicAddDouble(&task->hab_block[i * task->nsgfa + j], block_val);
        }
      }
    }
  }
  __syncthreads(); // TODO: not really neded because of concurrent writes to
  // cab
}

/*******************************************************************************
 * \brief Cuda kernel for integrating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void integrate_kernel(const kernel_params params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  fill_smem_task(&params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params.smem_cab_offset];
  double *smem_alpha = &shared_memory[params.smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params.smem_cxyz_offset];

  memset(smem_cxyz, 0, ncoset(task.lp) * sizeof(double));
  __syncthreads();

  cxyz_to_grid(&params, &task, smem_cxyz, params.grid);

  memset(smem_cab, 0, task.n1 * task.n2 * sizeof(double));
  __syncthreads();

  compute_alpha(&params, &task, smem_alpha);
  cab_to_cxyz(&params, &task, smem_alpha, smem_cab, smem_cxyz);

  store_hab(&params, &task, smem_cab);

  // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
  //  printf("la_min: %i %lb_min: %i, ncoset: %i, %i \n",
  //      task.la_min, task.lb_min,
  //      ncoset(task.la_min-1), ncoset(task.lb_min-1));
  //  //    for (int k = 0; k < task.ncosetb; k++) {
  //  //      for (int l = 0; l < task.ncoseta; l++) {
  //  //          printf("cab %i %i %le\n", k, l, smem_cab[k * task.ncoseta +
  //  l]);
  //  //      }
  //  //    }
  //  // printf("cxyz %i %i %le\n",0, 0, smem_cxyz[0]);
  //}
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that integrates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_integrate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool orthorhombic, const bool compute_tau,
    const bool calculate_forces, const int npts_global[3],
    const int npts_local[3], const int shift_local[3],
    const int border_width[3], const double dh[3][3], const double dh_inv[3][3],
    const cudaStream_t stream, const double *pab_blocks_dev,
    const double *grid_dev, double *hab_blocks_dev, double *forces_dev,
    double *virial_dev) {

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do.
  }

  init_constant_memory();

  // Compute required shared memory.
  // TODO: Currently, cab's indicies run over 0...ncoset[lmax],
  //       however only ncoset(lmin)...ncoset(lmax) are actually needed.
  const process_ldiffs ldiffs =
      process_get_ldiffs(calculate_forces, calculate_forces, compute_tau);
  const int la_max = task_list->lmax + ldiffs.la_max_diff;
  const int lb_max = task_list->lmax + ldiffs.lb_max_diff;
  const int lp_max = la_max + lb_max;
  const int cab_len = ncoset(lb_max) * ncoset(la_max);
  const int alpha_len = 3 * (lb_max + 1) * (la_max + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const size_t smem_per_block =
      (cab_len + alpha_len + cxyz_len) * sizeof(double);

  if (smem_per_block > 48 * 1024) {
    fprintf(stderr, "ERROR: Not enough shared memory.\n");
    fprintf(stderr, "cab_len: %i, ", cab_len);
    fprintf(stderr, "alpha_len: %i, ", alpha_len);
    fprintf(stderr, "cxyz_len: %i, ", cxyz_len);
    fprintf(stderr, "total smem_per_block: %f kb\n\n", smem_per_block / 1024.0);
    abort();
  }

  // assert(compute_tau == false);
  // assert(calculate_forces == false);

  // kernel parameters
  kernel_params params;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.orthorhombic = orthorhombic;
  params.compute_tau = compute_tau;
  params.calculate_forces = calculate_forces;
  params.grid = grid_dev;
  params.tasks = task_list->tasks_dev;
  params.atom_kinds = task_list->atom_kinds_dev;
  params.basis_sets = task_list->basis_sets_dev;
  params.block_offsets = task_list->block_offsets_dev;
  params.atom_positions = task_list->atom_positions_dev;
  params.pab_blocks = pab_blocks_dev;
  params.hab_blocks = hab_blocks_dev;
  params.forces = forces_dev;
  params.virial = virial_dev;
  params.la_min_diff = ldiffs.la_min_diff;
  params.lb_min_diff = ldiffs.lb_min_diff;
  params.la_max_diff = ldiffs.la_max_diff;
  params.lb_max_diff = ldiffs.lb_max_diff;
  memcpy(params.dh, dh, 9 * sizeof(double));
  memcpy(params.dh_inv, dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, shift_local, 3 * sizeof(int));
  memcpy(params.border_width, border_width, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 8, 8);

  integrate_kernel<<<nblocks, threads_per_block, smem_per_block, stream>>>(
      params);
}

#endif // __GRID_CUDA
// EOF
